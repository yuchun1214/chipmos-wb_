#include <include/population.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void initializePopulation(population_t * pop){
    
    int AMOUNT_OF_R_CHROMOSOMES = pop->parameters.AMOUNT_OF_R_CHROMOSOMES;
    int AMOUNT_OF_JOBS = pop->task.AMOUNT_OF_JOBS;
    int AMOUNT_OF_MACHINES = pop->task.AMOUNT_OF_MACHINES;

    job_t ** jobs;
    job_t ** address_of_jobs;
    job_t * job_entry;
    
    // malloc for jobs;
    cudaCheck(hipHostMalloc((void**)&address_of_jobs, sizeof(job_t*)*AMOUNT_OF_R_CHROMOSOMES), "hipHostMalloc for address_of_jobs");
    // malloc for entry
    for(int i = 0; i < AMOUNT_OF_R_CHROMOSOMES; ++i){
        cudaCheck(hipMalloc((void**)&job_entry, sizeof(job_t)*AMOUNT_OF_JOBS), "hipMalloc for entry of jobs");
        cudaCheck(hipMemcpy(job_entry, pop->task.jobs, AMOUNT_OF_JOBS, hipMemcpyHostToDevice), "hipMemcpy entry of jobs from host to device");
        address_of_jobs[i] = job_entry;
    }
    cudaCheck(hipMalloc((void**)&jobs, sizeof(job_t *)*AMOUNT_OF_R_CHROMOSOMES), "hipMalloc for jobs");
    cudaCheck(hipMemcpy(jobs, address_of_jobs, sizeof(job_t*)*AMOUNT_OF_R_CHROMOSOMES, hipMemcpyHostToDevice), "hipMemcpy jobs from host to device");
    
   
}

