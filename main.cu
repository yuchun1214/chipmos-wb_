#include "hip/hip_runtime.h"
#include <ctime>
#include <exception>
#include <initializer_list>


#include <iostream>
#include <iterator>
#include <map>
#include <set>
#include <stdexcept>
#include <string>
#include <system_error>
#include "include/chromosome_base.h"
#include "include/job_base.h"
#include "include/linked_list.h"
#include "include/machine_base.h"

#include "include/arrival.h"
#include "include/chromosome.h"
#include "include/condition_card.h"
#include "include/csv.h"
#include "include/da.h"
#include "include/entity.h"
#include "include/infra.h"
#include "include/lot.h"
#include "include/population.h"
#include "include/route.h"

#include <hip/hip_runtime.h>
#include <pthread.h>


using namespace std;

// round_t createARound(vector<lot_group_t> group, machines_t &machines,
// ancillary_resources_t & tools, ancillary_resources_t & wires);
//
//
// void initializeARound(round_t * r);
//
// void initializePopulation(population_t *pop, machines_t & machines,
// ancillary_resources_t & tools, ancillary_resources_t & wires);
//
//
// void geneticAlgorithm(population_t * pop);


task_t createTaskFromLotGroups(vector<lot_group_t> groups,
                               ancillary_resources_t &tools,
                               ancillary_resources_t &wires,
                               machines_t &machines);

int main(int argc, const char *argv[])
{
    
    // need a config file
    csv_t config_csv(argv[1], "r", true, true);
    map<string, string> elements = config_csv.getElements(0);

    csv_t lot_csv(elements["nlots"], "r", true, true);
    lot_csv.setHeaders(
        map<string, string>({{"route", "route"},
                             {"lot_number", "lot_number"},
                             {"pin_package", "pin_package"},
                             {"bd_id", "recipe"},
                             {"prod_id", "prod_id"},
                             {"part_id", "part_id"},
                             {"part_no", "part_no"},
                             {"urgent_code", "urgent_code"},
                             {"qty", "qty"},
                             {"dest_oper", "dest_oper"},
                             {"oper", "dest_oper"},
                             {"hold", "hold"},
                             {"mvin", "mvin"},
                             {"queue_time", "queue_time"},
                             {"fcst_time", "fcst_time"},
                             {"amount_of_tools", "amount_of_tools"},
                             {"amount_of_wires", "amount_of_wires"},
                             {"CAN_RUN_MODELS", "CAN_RUN_MODELS"},
                             {"PROCESS_TIME", "PROCESS_TIME"},
                             {"uphs", "uphs"},
                             {"customer", "customer"}}));
    lot_csv.trim(" ");
    vector<lot_t> allots;
    for (int i = 0, nrows = lot_csv.nrows(); i < nrows; ++i) {
        allots.push_back(lot_t(lot_csv.getElements(i)));
    }

    lots_t lots;
    lots.addLots(allots);
   

    ancillary_resources_t tools(lots.amountOfTools());
    ancillary_resources_t wires(lots.amountOfWires());

    csv_t machine_csv(elements["nmachines"], "r", true, true);
    machine_csv.trim(" ");
    machine_csv.setHeaders(map<string, string>({{"entity", "ENTITY"},
                                                {"model", "MODEL"},
                                                {"recover_time", "OUTPLAN"}}));

    csv_t location_csv(elements["locations"], "r", true, true);
    location_csv.trim(" ");
    location_csv.setHeaders(
        map<string, string>({{"entity", "Entity"}, {"location", "Location"}}));


    char *text = strdup(elements["std_time"].c_str());
    entities_t entities(text);
    entities.addMachines(machine_csv, location_csv);
    machines_t machines;
    machines.addMachines(entities.getAllEntity());

    vector<vector<lot_group_t> > round_groups = lots.rounds(entities);

    // srand(time(NULL));
    double tm = 60; 
    double total_completion_time = 0;
    for(int i = 0; i < round_groups.size(); ++i){
        task_t t = createTaskFromLotGroups(round_groups[i], tools, wires, machines);
        // printf("amount of lots = %d\n", t.AMOUNT_OF_JOBS);
        // printf("amount of machines = %d\n", t.AMOUNT_OF_MACHINES);

        population_t pop = {.no = 0,
                            .parameters = {.AMOUNT_OF_CHROMOSOMES = 50,
                                           .AMOUNT_OF_R_CHROMOSOMES = 100,
                                           .EVOLUTION_RATE = 0.8,
                                           .SELECTION_RATE = 0.2,
                                           .GENERATIONS = 60 / round_groups.size(),
                                           .SWAP_CHROMOSOMES = 60},
                            .task = t};
        initializePopulation(&pop);
        total_completion_time += geneticAlgorithm(&pop);
    }
    printf("%f\n", total_completion_time);

    // pthread_t thread;
    // pthread_create(&thread, NULL, geneticAlgorithm, &pop);
    // pthread_join(thread, NULL);

    // population_t populations[10];
    // pthread_t threads[10];

    // for(int i = 0; i < 10; ++i){
    //     populations[i] = population_t{
    //         .no = (unsigned)i,
    //         .parameters = {
    //             .AMOUNT_OF_CHROMOSOMES = 50,
    //             .AMOUNT_OF_R_CHROMOSOMES = 100,
    //             .EVOLUTION_RATE = 0.8,
    //             .SELECTION_RATE = 0.2,
    //             .GENERATIONS = 20,
    //             .SWAP_CHROMOSOMES = 10
    //         },
    //         .task = t,
    //     };
    // }
    //
    // for(int i = 0; i < 10; ++i){
    //     initializePopulation(&populations[i]);
    // }
    //
    // clock_t t1;
    // clock_t t2;
    //
    // for(int i = 0; i < 3; ++i){
    //     t1 = clock();
    //     t2 = t1 + 60 * CLOCKS_PER_SEC;
    //     for(int i = 0; i < 10; ++i){
    //         pthread_create(&threads[i], NULL,  geneticAlgorithm,
    //         (void*)&populations[i]);
    //     }

    //     for(int i = 0; i < 10; ++i){
    //         pthread_join(threads[i], NULL);
    //     }
    //     swapPopulation(populations, 10);
    // }
    //
    // double bestFitnessValue = 1000000000;
    // for(int i = 0; i < 10; ++i){
    //     for(int j = 0; j <  populations[i].parameters.SWAP_CHROMOSOMES; ++j){
    //         if(populations[i].chromosomes.host_chromosomes[j].fitnessValue <
    //         bestFitnessValue){
    //             bestFitnessValue =
    //             populations[i].chromosomes.host_chromosomes[j].fitnessValue;
    //         }
    //     }
    // }

    // FILE * file = fopen("result.txt", "a+");
    // fprintf(file, "%f\n", bestFitnessValue);
    // fclose(file);

    return 0;
}


task_t createTaskFromLotGroups(vector<lot_group_t> groups,
                               ancillary_resources_t &tools,
                               ancillary_resources_t &wires,
                               machines_t &machines)
{
    // setup jobs
    int AMOUNT_OF_JOBS = 0;
    int AMOUNT_OF_MACHINES = 0;
    int k = 0;
    // vector<lot_group_t> ngroups(groups.begin() + 1, groups.begin() + 8);
    // groups = ngroups;
    iter(groups, i)
    {
        AMOUNT_OF_JOBS += groups[i].lots.size();
        AMOUNT_OF_MACHINES += groups[i].machine_amount;
    }

    job_t *jobs;
    cudaCheck(hipHostMalloc(&jobs, sizeof(job_t) * AMOUNT_OF_JOBS),
              "cudasMallocHost for jobs");

    // setup jobs data
    iter(groups, i)
    {
        iter(groups[i].lots, j)
        {
            jobs[k] = groups[i].lots[j]->job();
            ++k;
        }
    }

    // setup process time
    int *size_of_process_times;
    cudaCheck(
        hipHostMalloc(&size_of_process_times, sizeof(int) * AMOUNT_OF_JOBS),
        "hipHostMalloc for size_of_process_times");
    process_time_t **process_times;
    cudaCheck(hipHostMalloc(&process_times,
                             sizeof(process_time_t *) * AMOUNT_OF_JOBS),
              "hipHostMalloc for process times");
    process_time_t *process_times_entry;
    k = 0;
    iter(groups, i)
    {
        iter(groups[i].lots, j)
        {
            map<string, double> mpts =
                groups[i].lots[j]->getEntitiyProcessTime();

            cudaCheck(hipHostMalloc(&process_times_entry,
                                     sizeof(process_time_t) * mpts.size()),
                      "hipHostMalloc for process time entry");
            int l = 0;
            for (map<string, double>::iterator it = mpts.begin();
                 it != mpts.end(); it++) {
                process_times_entry[l] = process_time_t{
                    .machine_no = convertEntityNameToUInt(it->first),
                    .process_time = it->second};
                ++l;
            }
            process_times[k] = process_times_entry;
            size_of_process_times[k] = mpts.size();
            jobs[k].base.size_of_process_time = mpts.size();
            ++k;
        }
    }

    // setup machines
    map<string, machine_t *> allmachines = machines.getMachines();
    machine_t *ms;
    cudaCheck(hipHostMalloc(&ms, sizeof(machine_t) * AMOUNT_OF_MACHINES),
              "hipHostMalloc for machines");
    k = 0;
    iter(groups, i)
    {
        iter(groups[i].entities, j)
        {
            string ent_name = groups[i].entities[j]->entity_name;
            ms[k] = *(allmachines[ent_name]);
            ++k;
        }
    }

    // setup tools and machines
    tool_t *ts;
    wire_t *ws;
    cudaCheck(hipHostMalloc(&ts, sizeof(tool_t) * AMOUNT_OF_MACHINES),
              "hipHostMalloc for tools");
    cudaCheck(hipHostMalloc(&ws, sizeof(wire_t) * AMOUNT_OF_MACHINES),
              "hipHostMalloc for wires");
    k = 0;
    iter(groups, i)
    {
        vector<tool_t *> v_ts =
            tools.aRound(groups[i].tool_name, groups[i].machine_amount);
        vector<wire_t *> v_ws =
            wires.aRound(groups[i].wire_name, groups[i].machine_amount);
        for (int j = 0; j < groups[i].machine_amount; ++j) {
            ts[k] = *(v_ts[j]);
            ws[k] = *(v_ws[j]);
            ++k;
        }
    }
    return task_t{.AMOUNT_OF_JOBS = AMOUNT_OF_JOBS,
                  .AMOUNT_OF_MACHINES = AMOUNT_OF_MACHINES,
                  .jobs = jobs,
                  .machines = ms,
                  .tools = ts,
                  .wires = ws,
                  .process_times = process_times,
                  .size_of_process_times = size_of_process_times};
}
